#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>

// CUDA kernel for element-wise computation: c[i] = sqrt(a[i]) + log(b[i])
__global__ void sqrt_log_kernel(const float* a, const float* b, float* c, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        c[i] = sqrtf(a[i]) + logf(b[i] + 1e-6f);
    }
}

// Host function that manages GPU memory and timing, and launches the kernel
extern "C" {
    void gpu_math(const float* a, const float* b, float* c, int N, float* kernel_time_ms) {
        float* d_a, * d_b, * d_c;
        size_t size = N * sizeof(float);

        // Allocate memory on device
        hipMalloc(&d_a, size);
        hipMalloc(&d_b, size);
        hipMalloc(&d_c, size);

        // Copy data from host to device
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (N + blockSize - 1) / blockSize;

        // Measure kernel execution time using CUDA events
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        sqrt_log_kernel << <numBlocks, blockSize >> > (d_a, d_b, d_c, N);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(kernel_time_ms, start, stop);

        // Copy results back to host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        // Destroy CUDA events
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
}