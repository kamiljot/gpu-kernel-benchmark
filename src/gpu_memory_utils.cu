#include "hip/hip_runtime.h"
// Implements device memory allocation, deallocation, and copy routines for GPU arrays.

#include "gpu_memory_utils.h"

void allocate_and_copy(const float* a, const float* b, float** d_a, float** d_b, float** d_c, int N) {
    size_t size = N * sizeof(float);
    hipMalloc(d_a, size);
    hipMalloc(d_b, size);
    hipMalloc(d_c, size);
    hipMemcpy(*d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(*d_b, b, size, hipMemcpyHostToDevice);
}

void allocate_and_copy_vec4(const float* a, const float* b, float4** d_a4, float4** d_b4, float4** d_c4, int N_vec4) {
    size_t size = N_vec4 * sizeof(float4);
    hipMalloc(d_a4, size);
    hipMalloc(d_b4, size);
    hipMalloc(d_c4, size);
    hipMemcpy(*d_a4, a, size, hipMemcpyHostToDevice);
    hipMemcpy(*d_b4, b, size, hipMemcpyHostToDevice);
}

void free_device(float* d_a, float* d_b, float* d_c) {
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
