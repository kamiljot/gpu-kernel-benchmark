#include "hip/hip_runtime.h"
// Host launchers for different {{name}} kernel variants.

#include "{{name}}.h"
#include "{{name}}_kernels.cuh"
#include "cuda_utils.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

extern "C" float run_{ {name} }_global(const float* a, const float* b, float* c, int N) {
    float* d_a, * d_b, * d_c;
    hipEvent_t start, stop;
    float ms = -1.0f;

    try {
        std::tie(d_a, d_b, d_c) = allocate_and_copy_to_device(a, b, N);

        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));

        int blockSize = 256;
        int gridSize = (N + blockSize - 1) / blockSize;

        CHECK_CUDA(hipEventRecord(start));
        { { name } }_global_kernel << <gridSize, blockSize >> > (d_a, d_b, d_c, N);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipEventRecord(stop));

        CHECK_CUDA(hipEventSynchronize(stop));
        CHECK_CUDA(hipEventElapsedTime(&ms, start, stop));

        copy_from_device_and_free(c, d_c, d_a, d_b, N);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    catch (const std::exception& e) {
        std::cerr << "CUDA error in run_{{name}}_global: " << e.what() << std::endl;
        return -1.0f;
    }
    return ms;
}

extern "C" float run_{ {name} }_shared(const float* a, const float* b, float* c, int N) {
    float* d_a, * d_b, * d_c;
    hipEvent_t start, stop;
    float ms = -1.0f;

    try {
        std::tie(d_a, d_b, d_c) = allocate_and_copy_to_device(a, b, N);

        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));

        int blockSize = 256;
        int gridSize = (N + blockSize - 1) / blockSize;
        size_t sharedMemSize = 2 * blockSize * sizeof(float);

        CHECK_CUDA(hipEventRecord(start));
        { { name } }_shared_kernel << <gridSize, blockSize, sharedMemSize >> > (d_a, d_b, d_c, N);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipEventRecord(stop));

        CHECK_CUDA(hipEventSynchronize(stop));
        CHECK_CUDA(hipEventElapsedTime(&ms, start, stop));

        copy_from_device_and_free(c, d_c, d_a, d_b, N);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    catch (const std::exception& e) {
        std::cerr << "CUDA error in run_{{name}}_shared: " << e.what() << std::endl;
        return -1.0f;
    }
    return ms;
}

extern "C" float run_{ {name} }_float4(const float* a, const float* b, float* c, int N) {
    int N_vec4 = N / 4;

    float4* d_a4, * d_b4, * d_c4;
    hipEvent_t start, stop;
    float ms = -1.0f;

    try {
        std::tie(d_a4, d_b4, d_c4) = allocate_and_copy_to_device_float4(a, b, N_vec4);

        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));

        int blockSize = 256;
        int gridSize = (N_vec4 + blockSize - 1) / blockSize;

        CHECK_CUDA(hipEventRecord(start));
        { { name } }_float4_kernel << <gridSize, blockSize >> > (d_a4, d_b4, d_c4, N_vec4);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipEventRecord(stop));

        CHECK_CUDA(hipEventSynchronize(stop));
        CHECK_CUDA(hipEventElapsedTime(&ms, start, stop));

        copy_from_device_and_free_float4(c, d_c4, d_a4, d_b4, N_vec4);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    catch (const std::exception& e) {
        std::cerr << "CUDA error in run_{{name}}_float4: " << e.what() << std::endl;
        return -1.0f;
    }
    return ms;
}